// %%cuda --name hello.cu


#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
using namespace std;



__device__ float sine(int x){
    float INV_ROOT_TWO= (1.0f)/sqrt(2.0f);
    if(x==0){
        return 0;
    }
    else if(x==45){
        return INV_ROOT_TWO;
    }
    else{
        return  -1*INV_ROOT_TWO;
    }
}


__device__ float cosine(int x){
    float INV_ROOT_TWO= (1.0f)/sqrt(2.0f);
    if(x==0){
        return 1;
    }
    else{
        return INV_ROOT_TWO;
    }
}



__device__ void get_new_coord(float* point,float* pivot,float* new_pt,int theta){  //[0] is x-coord and [1] is y-coord
    //left corner is origin rightwards +ve x-axis and top +ve y-axis 
    new_pt[0]=pivot[0]+(point[0]-pivot[0])*cosine(theta)-(point[1]-pivot[1])*sine(theta);
    new_pt[1]=pivot[1]+(point[0]-pivot[0])*sine(theta)+(point[1]-pivot[1])*cosine(theta);
    return;    


}




__device__ void get_bound_pos(int r,int c,int theta,int *bdrs){

    float m1=(float) c;
    float n1=(float) r;



    float a1[2]={m1,0.0f}; //bottom right
    float a2[2]={m1,n1}; //top right
    float a3[2]={0.0f,n1};  //top left
    float P[2]={0.0f,0.0f};

    float p1[2];
    float p2[2];
    float p3[3];


    get_new_coord(a1,P,p1,theta); //new pos of bottom right
    get_new_coord(a2,P,p2,theta); //new pos of top right
    get_new_coord(a3,P,p3,theta); //new pos of top left

    int lx,rx,ty,by;  //left-x,right-x,top-y,bottom-y

    if(theta==0){
        lx=0;
        rx=c;
        ty=r;
        by=0;

    }
    else if(theta==45){
        lx=ceil(p3[0]);
        rx=floor(p1[0]);
        ty=floor(p2[1]);
        by=0;


    }
    else if(theta==-45){
        lx=0;
        rx=floor(p2[0]);
        ty=floor(p3[1]);
        by=ceil(p1[1]);


    }

    bdrs[0]=lx;
    bdrs[1]=rx;
    bdrs[2]=ty;
    bdrs[3]=by;



    return;
  

}


__device__ bool check_in_limits(float* Grey_img,int* bounds,int d_m,int d_n,int i,int j,float th2_val,float avg_val,float* d_A){

    int l1=d_m;
    int l2=d_n;

    int nl1=bounds[1]-bounds[0]+1;
    int nl2=bounds[2]-bounds[3]+1;
    float val=0.0f;

    if(j+bounds[0]<0 || j+bounds[1]>=l2 || i+bounds[2]>=l1 || i+bounds[3]<0){
        return false;
    }

    for(int u=i+bounds[3];u<=i+bounds[2];u++){
        for(int v=j+bounds[0];v<=j+bounds[1];v++){
           
            val+=Grey_img[u*d_n+v];

        }

        
    }
    val=(val)/(float)(nl1*nl2);
  
    if(abs(val-avg_val)<th2_val){
        d_A[i*d_n+j]=val-avg_val;
        //cout<<i<<" "<<j<<" "<<val<<"\n";
        return true;
    }
    d_A[i*d_n+j]=-1.0f;
    return false;


}

__device__ bool interpolate(float* point,float* val,int* D,int d_m,int d_n){ //vector<float> &point,vector<vector<vector<int>>> &D){
    int x1=floor(point[0]);
    int y1=floor(point[1]);
    int x2=1+floor(point[0]);
    int y2=1+floor(point[1]);
    
    

     if(x1<0 || y1<0 || x2>=d_n || y2>=d_m){
        return false;

     }

   

    //z00*(1-x)*(1-y) + z10*x*(1-y) + z01*(1-x)*y + z11*x*y

    float z00;
    float z10;
    float z01;
    float z11;



      
    
    for(int u=0;u<3;u++){
       
        z00=(float) D[y1*3*d_n+x1*3+u];
        z01=(float) D[y2*3*d_n+x1*3+u];
        z10=(float) D[y1*3*d_n+x2*3+u];
        z11=(float) D[y2*3*d_n+x2*3+u];

        val[u]=z00*(x2-point[0])*(y2-point[1])+z01*(point[1]-y1)*(x2-point[0])+z10*(point[0]-x1)*(y2-point[1])+z11*(point[0]-x1)*(point[1]-y1);
    }



    return true;
}


__device__ void get_best_picks(int* D,int* Q,int i,int j,float th1_val,int theta,int q_m,int q_n,int d_m,int d_n,int top_n,float* d_R){


    float point[2];
    float pivot[2];
    float new_pos[2];
    float interpol_val[4];

    point[0]=0.0f;
    point[1]=0.0f;

    pivot[0]=(float)j;
    pivot[1]=(float)i;
    

    float rmsd=0.0f;
    float temp;
    float f;
    bool pol_flag;

    for(int u=i;u<i+q_m;u++){
        for(int v=j;v<j+q_n;v++){
            point[0]=(float)v;
            point[1]=(float)u;
            get_new_coord(point,pivot,new_pos,theta);
            pol_flag=interpolate(new_pos,interpol_val,D,d_m,d_n);
            if(pol_flag==false){
                d_R[i*d_n+j]=-1.0f;
                return;
            }
            for(int w=0;w<3;w++){
                temp=(Q[(u-i)*3*q_n+(v-j)*3+w]-interpol_val[w]);
                rmsd+=temp*temp;

            }

           

        }


    }

    float n1=(float) (3*q_m*q_n);
    rmsd/=n1;
    f=sqrt(rmsd);
    //cout<<i<<" "<<j<<" "<<theta<<" "<<f<<"\n";
    if(f<th1_val){
        d_R[i*d_n+j]=f;
        //cout<<i<<" "<<j<<" "<<theta<<" "<<"\n";
        return;

    }
    d_R[i*d_n+j]=-1.0f;


   
    return;
   
}
   







__global__ void filtering(float* Grey_img,int theta,float th2_val,float avg_val,int q_m,int q_n,int d_m,int d_n,
int* D,int* Q,float th1_val,int top_n,float* d_A,float* d_R){

    int bounds[4];
    get_bound_pos(q_m,q_n,theta,bounds);
    int i=blockIdx.y;
    int j=blockIdx.x;


    bool filt_flag=check_in_limits(Grey_img,bounds,d_m,d_n,i,j,th2_val,avg_val,d_A);
    if(filt_flag){
       get_best_picks(D,Q,i,j,th1_val,theta,q_m,q_n,d_m,d_n,top_n,d_R);              

    }
    else{
        d_R[i*d_n+j]=-1.0f;
    }
    

    return;


}







int main(int argc,char* argv[]){

    auto beg=std::chrono::high_resolution_clock::now();

    string data_img_path=argv[1];
    string query_img_path=argv[2];
    float th_1=stof(argv[3]);
    float th_2=stof(argv[4]);
    int n=stoi(argv[5]);

    

    int q_m,q_n;
    int d_m,d_n;

    int s=0;
    float avg_val;
    int val;

    
    ifstream fin;
   
    fin.open(data_img_path);
   
    fin>>d_m>>d_n;
    int* D=new int[3*d_m*d_n];
    
    for(int i=0;i<d_m;i++){
        for(int j=0;j<d_n;j++){
            for(int k=0;k<3;k++){
                fin>>val;
                D[(d_m-1-i)*d_n*3+j*3+k]=val;
            }
        }

    }
  
    fin.close();
    fin.open(query_img_path);
    cout<<query_img_path<<"\n";
   
    fin>>q_m>>q_n;
    int* Q=new int[3*q_m*q_n];
      
    for(int i=0;i<q_m;i++){
        for(int j=0;j<q_n;j++){
            for(int k=0;k<3;k++){
                fin>>val;
                Q[(q_m-1-i)*q_n*3+j*3+k]=val;
                s+=val;
            }
        }

    }
    
    fin.close();

   
    avg_val=(float)(s)/(float)(3*q_m*q_n);

    float ans=0.0f;
    float* G=new float[d_m*d_n];

    for(int i=0;i<d_m*d_n;i++){
        ans=(float)(D[3*i]+D[3*i+1]+D[3*i+2])/(3.0f);
        G[i]=ans;
    }
    
    
    cout<<q_m<<" "<<q_n<<" "<<d_m<<" "<<d_n<<"\n";


    int *d_Q;
    int *d_D;
    float *d_G;

    float* d_A;
    float* d_R;

    float* A=new float[d_m*d_n];
    float* R=new float[d_m*d_n];




    hipMalloc((void**)&d_Q,sizeof(float)*3*q_m*q_n);
    hipMalloc((void**)&d_D,sizeof(float)*3*d_m*d_n);
    hipMalloc((void**)&d_G,sizeof(float)*d_m*d_n);


    hipMalloc((void**)&d_A,sizeof(float)*d_m*d_n);
    hipMalloc((void**)&d_R,sizeof(float)*d_m*d_n);


    hipMemcpy(d_D, D,sizeof(float)*3*d_m*d_n,hipMemcpyHostToDevice);
    hipMemcpy(d_G, G, sizeof(float)*d_m*d_n, hipMemcpyHostToDevice);
    hipMemcpy(d_Q, Q, sizeof(float)*3*q_m*q_n, hipMemcpyHostToDevice);


    delete D;
    delete G;
    delete Q;



    
    


    priority_queue<pair<float,tuple<int,int,int,float>>>PQ;
    vector<pair<float,tuple<int,int,int,float>>>V;
    vector<int>angles{-45,0,45};

    dim3 grid(d_n,d_m);

    for(auto h:angles){
        filtering<<<grid,1>>>(d_G,h,th_2,avg_val,q_m,q_n,d_m,d_n,d_D,d_Q,th_1,n,d_A,d_R);
        hipMemcpy(A,d_A, sizeof(float)*d_m*d_n, hipMemcpyDeviceToHost);
        hipMemcpy(R,d_R, sizeof(float)*d_m*d_n, hipMemcpyDeviceToHost);
       

        for(int i=0;i<d_m;i++){
            for(int j=0;j<d_n;j++){
                if(R[i*d_n+j]>=0.0f){
                    pair<float,tuple<int,int,int,float>>P;
                    P.first=R[i*d_n+j];
                    get<0>(P.second)=i;
                    get<1>(P.second)=j;
                    get<2>(P.second)=h;
                    get<3>(P.second)=A[i*d_n+j];
                    PQ.push(P);
                    if(PQ.size()>n){
                        PQ.pop();
                    }
                }
            }
        }
    }


    
    delete A;
    delete R;

    hipFree(d_D);
    hipFree(d_G);
    hipFree(d_Q);
    hipFree(d_A);
    hipFree(d_R);

    while(!PQ.empty()){
         auto x=PQ.top();
         V.push_back(x);
         //cout<<x.first<<" "<<get<3>(x.second)<<" "<<get<0>(x.second)<<" "<<get<1>(x.second)<<" "<<get<2>(x.second)<<"\n";
         PQ.pop();
    }

    ofstream fout;
    fout.open("output.txt");
    for(int i=V.size()-1;i>=0;i--){
        auto x=V[i];
        fout<<get<0>(x.second)<<" "<<get<1>(x.second)<<" "<<get<2>(x.second)<<"\n";
    }

    fout.close();

    auto end=std::chrono::high_resolution_clock::now();
    
    cout<<"Time taken for execution :"<<(1e-6*(std::chrono::duration_cast<std::chrono::nanoseconds>(end-beg)).count())<<" ms"<<"\n";
    

   
   

}